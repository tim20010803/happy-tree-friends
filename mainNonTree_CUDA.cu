#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>
#include <omp.h>
#include <random>


// Define particle structure
struct Particle {
    float posi[2];
    float velocity[2];
    float acceleration[2];
    float mass;
};

// Calculate system momentum
std::vector<float> calculate_system_momentum(const std::vector<Particle>& particles) {
    float total_momentum_x = 0.0;
    float total_momentum_y = 0.0;

    for (const Particle& p : particles) {
        total_momentum_x += p.mass * p.velocity[0];
        total_momentum_y += p.mass * p.velocity[1];
    }

    return { total_momentum_x, total_momentum_y };
}

// Calculate system energy
float calculate_system_energy(const std::vector<Particle>& particles, float G) {
    float total_energy = 0.0;

    for (const Particle& p : particles) {
        float kinetic_energy = 0.5 * p.mass * (p.velocity[0] * p.velocity[0] + p.velocity[1] * p.velocity[1]);
        float potential_energy = 0.0;

        for (const Particle& other : particles) {
            if (&p != &other) {
                float dx = other.posi[0] - p.posi[0];
                float dy = other.posi[1] - p.posi[1];
                float dist = sqrt(dx * dx + dy * dy);
                potential_energy -= G * p.mass * other.mass / dist;
            }
        }

        total_energy += kinetic_energy + potential_energy;
    }

    return total_energy;
}



__global__ void calculate_gravity_cuda_kernel(Particle* particles, float G, int num_particles) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p1 = particles[index];

        for (int j = 0; j < num_particles; j++) {
            if (j != index) {
                Particle& p2 = particles[j];
                float dx = p2.posi[0] - p1.posi[0];
                float dy = p2.posi[1] - p1.posi[1];
                float dist_squared = dx * dx + dy * dy;
                float dist_cubed = dist_squared * sqrt(dist_squared);

                float force_magnitude = G * p1.mass * p2.mass / dist_cubed;
                float force_x = force_magnitude * dx;
                float force_y = force_magnitude * dy;

                atomicAdd(&p1.acceleration[0], force_x / p1.mass);
                atomicAdd(&p1.acceleration[1], force_y / p1.mass);


            }
        }
    }
}

__global__ void Verlet_velocity_cuda_kernel(Particle* particles, float G, float dt, int num_particles) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p = particles[index];

        p.velocity[0] += p.acceleration[0] * dt;
        p.velocity[1] += p.acceleration[1] * dt;

        p.posi[0] += p.velocity[0] * dt;
        p.posi[1] += p.velocity[1] * dt;

        p.acceleration[0] = 0.0;
        p.acceleration[1] = 0.0;
    }
}

int main() {
    // Define simulation parameters
    const float G = 6.674e-11;
    const float M_PI = 3.14159;

    std::cout << std::fixed << std::setprecision(12);

    // Input time and time step
    float t = 4.0 * M_PI / sqrt(G * 1000000000);
    float dt = 0.001;

    // Perform simulation
    int num_steps = t / dt;

// Random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist_pos(-1.0f, 1.0f);
    std::uniform_real_distribution<float> dist_vel(-10.0f, 10.0f);
    std::uniform_real_distribution<float> dist_mass(1.0e6f, 1.0e7f);

    // Generate particles
    std::vector<Particle> particles;
    particles.reserve(10000);
    for (int i = 0; i < 10000; ++i) {
        Particle particle;
        particle.posi[0] = dist_pos(gen);
        particle.posi[1] = dist_pos(gen);
        particle.velocity[0] = dist_vel(gen);
        particle.velocity[1] = dist_vel(gen);
        particle.acceleration[0] = 0.0f;
        particle.acceleration[1] = 0.0f;
        particle.mass = dist_mass(gen);
        particles.push_back(particle);
    }


    float start_time = omp_get_wtime();

    // Perform Verlet simulation using CUDA
    Particle* d_particles;
    size_t particlesSize = particles.size() * sizeof(Particle);

    hipMalloc((void**)&d_particles, particlesSize);
    hipMemcpy(d_particles, particles.data(), particlesSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (particles.size() + blockSize - 1) / blockSize;

    for (int i = 0; i <= num_steps; i++) {
        calculate_gravity_cuda_kernel<<<gridSize, blockSize>>>(d_particles, G, particles.size());
        Verlet_velocity_cuda_kernel<<<gridSize, blockSize>>>(d_particles, G, dt, particles.size());
    }

    hipMemcpy(particles.data(), d_particles, particlesSize, hipMemcpyDeviceToHost);
    hipFree(d_particles);

    float end_time = omp_get_wtime();

    std::vector<float> system_momentum = calculate_system_momentum(particles);
    float system_energy = calculate_system_energy(particles, G);

    std::cout << "Verlet_velocity Time: " << (num_steps) * dt << std::endl;
    std::cout << "System Momentum (X, Y): (" << system_momentum[0] << ", " << system_momentum[1] << ")" << std::endl;
    std::cout << "System Energy: " << system_energy << std::endl;
    std::cout << "Total Execution Time: " << end_time - start_time << " seconds" << std::endl;

    return 0;
}
