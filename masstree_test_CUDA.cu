
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>
#include <omp.h>
#include <random>
#include <chrono>
#include <fstream>
#include <string>
#include <sstream>
#define G_CONST 6.67428e-11
// Define particle structure
struct Particle {
    float posi[2];
    float velocity[2];
    float acceleration[2];
    float acceleration_prev[2];
    float mass;
};

// 自定义reduction运算符：向量相加
#pragma omp declare reduction(vec_float_plus : std::vector<float> : \
    std::transform(omp_out.begin(), omp_out.end(), omp_in.begin(), omp_out.begin(), std::plus<float>())) \
    initializer(omp_priv = decltype(omp_orig)(omp_orig.size(), 0.0))

// 计算系统动量
std::vector<float> calculate_system_momentum(const std::vector<Particle>& particles) {
    std::vector<float> system_momentum(2, 0.0);

    #pragma omp parallel for reduction(vec_float_plus:system_momentum)
    for (int i = 0; i < particles.size(); ++i) {
        const Particle& particle = particles[i];
        system_momentum[0] += particle.mass * particle.velocity[0];
        system_momentum[1] += particle.mass * particle.velocity[1];
    }

    return system_momentum;
}

// 计算系统角动量
std::vector<float> calculate_system_angular_momentum(const std::vector<Particle>& particles) {
    std::vector<float> system_angular_momentum(1, 0.0);

    #pragma omp parallel for reduction(vec_float_plus:system_angular_momentum)
    for (int i = 0; i < particles.size(); ++i) {
        const Particle& particle = particles[i];
        system_angular_momentum[0] += particle.mass * (particle.posi[0] * particle.velocity[1] - particle.posi[1] * particle.velocity[0]);
    }

    return system_angular_momentum;
}


float calculate_system_energy(const std::vector<Particle>& particles, float G) {
    float total_kinetic_energy = 0.0;
    float total_potential_energy = 0.0;

    #pragma omp parallel for reduction(+:total_kinetic_energy, total_potential_energy)
    for (size_t i = 0; i < particles.size(); i++) {
        const auto& p = particles[i];

        // Calculate kinetic energy
        float speed_squared = p.velocity[0] * p.velocity[0] + p.velocity[1] * p.velocity[1];
        float kinetic_energy = 0.5 * p.mass * speed_squared;
        #pragma omp atomic
        total_kinetic_energy += kinetic_energy;

        // Calculate potential energy
        for (size_t j = 0; j < particles.size(); j++) {
            if (i == j) {
                continue;
            }

            const auto& other_p = particles[j];
            float dx = other_p.posi[0] - p.posi[0];
            float dy = other_p.posi[1] - p.posi[1];
            float distance = std::sqrt(dx * dx + dy * dy);
            float potential_energy = -G * p.mass * other_p.mass / distance;
            #pragma omp atomic
            total_potential_energy += potential_energy;
        }
    }

    return total_kinetic_energy + total_potential_energy;
}

__global__ void simulate_particles_cuda_kernel(Particle* particles, const float* particle_masses, float G, float dt, int num_particles) {
    extern __shared__ Particle sharedParticles[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p = particles[index];
        Particle& shared_p = sharedParticles[threadIdx.x];

        shared_p = p;

        __syncthreads();

        for (int j = 0; j < num_particles; j++) {
            if (j != index) {
                Particle& other = sharedParticles[j];
                float dx = other.posi[0] - shared_p.posi[0];
                float dy = other.posi[1] - shared_p.posi[1];
                float dist_squared = dx * dx + dy * dy;
                float dist_cubed = dist_squared * sqrt(dist_squared);

                float force_magnitude = G * shared_p.mass * particle_masses[j] / dist_cubed;
                float force_x = force_magnitude * dx;
                float force_y = force_magnitude * dy;

                atomicAdd(&shared_p.acceleration[0], force_x / shared_p.mass);
                atomicAdd(&shared_p.acceleration[1], force_y / shared_p.mass);
            }
        }

        p.posi[0] += p.velocity[0] * dt + 0.5 * p.acceleration[0] * dt * dt;
        p.posi[1] += p.velocity[1] * dt + 0.5 * p.acceleration[1] * dt * dt;
        p.velocity[0] += 0.5 * (p.acceleration[0] + p.acceleration_prev[0]) * dt;
        p.velocity[1] += 0.5 * (p.acceleration[1] + p.acceleration_prev[1]) * dt;
        p.acceleration_prev[0] = p.acceleration[0];
        p.acceleration_prev[1] = p.acceleration[1];

        p.acceleration[0] = 0.0f;
        p.acceleration[1] = 0.0f;
    }
}


int main() {
    // Define simulation parameters
    // const float G = 6.674e-11;

    // std::cout << std::fixed << std::setprecision(12);

    // // Input time and time step
    // float t = 0.001;
    // float dt = 0.001;

    // // Perform simulation
    // int num_steps = t / dt;
    // int particles_num = 100000;

    // Random number generator
    // std::random_device rd;
    // std::mt19937 gen(rd());
    // std::uniform_real_distribution<float> dist_pos(-1.0f, 1.0f);
    // std::uniform_real_distribution<float> dist_vel(-10.0f, 10.0f);
    // std::uniform_real_distribution<float> dist_mass(1.0e6f, 1.0e7f);

    // // Generate particles
    // std::vector<Particle> particles;
    // particles.reserve(particles_num);
    // for (int i = 0; i < particles_num; ++i) {
    //     Particle particle;
    //     particle.posi[0] = dist_pos(gen);
    //     particle.posi[1] = dist_pos(gen);
    //     particle.velocity[0] = dist_vel(gen);
    //     particle.velocity[1] = dist_vel(gen);
    //     particle.acceleration[0] = 0.0f;
    //     particle.acceleration[1] = 0.0f;
    //     particle.mass = dist_mass(gen);
    //     particles.push_back(particle);
    // }




    std::ifstream file("one_step_data.csv");
    std::vector<Particle> particles; // store the particles
    std::string line; // each line of the file data
    bool isFirstLine = true; // if line is the first line or not
    int particleNum =0;
    // to get all of the data
    while (std::getline(file, line)) {
        // delete the first line because the first line is
        // "Time,Particle,Mass,PositionX,PositionY,VectorX,VectorY,AccelerationX,AccelerationY"
        if (isFirstLine) {
            isFirstLine = false;
            continue; 
        }

        // set the parameter to store data
        std::istringstream iss(line);
        std::string element;
        std::vector<std::string> elements; 
        // change the line string into a vector
        while (std::getline(iss, element, ',')) {
            elements.push_back(element);
        }

        Particle particle;
        // input data
        double m = std::stod(elements[2]);
        double x = std::stod(elements[3]);
        double y = std::stod(elements[4]);
        double vx = std::stod(elements[5]);
        double vy = std::stod(elements[6]);
        double ax = std::stod(elements[7]);
        double ay = std::stod(elements[8]);
        // add the particle initial condition
        particle.mass = m;
        particle.posi[0]=(x);
        particle.posi[1]=(y);
        particle.velocity[0]=(vx);
        particle.velocity[1]=(vy);
        particle.acceleration[0]=(ax);
        particle.acceleration[1]=(ay);

        // add the particle
        particleNum++;
        particles.push_back(particle);
    }

    // int index = 1;
    // for (const auto& particle : particles) {        
    //     std::cout << "Particle: " << index << ", Mass: " << particle.mass 
    //     << ", PositionX: " << particle.posi[0] << ", PositionY: " << particle.posi[1] 
    //     << ", VectorX: " << particle.velocity[0] << ", VectorY: " << particle.velocity[1] 
    //     << ", AccelerationX: " << particle.acceleration[0] << ", AccelerationY: " << particle.acceleration[1] << std::endl;
    //     index++;
    // }
    double t=0.005 ,dt = 0.005;
    int num_steps = t / dt;
    std::cout << "Physical Time: " << (num_steps)*dt <<"seconds"<< std::endl;
    std::cout << particleNum <<  "particles"<< std::endl;
    std::cout << num_steps <<  "steps"<< std::endl;

    // Start timer
    auto start = std::chrono::high_resolution_clock::now();

    // Transfer particles to device memory
    Particle* d_particles;
    size_t particlesSize = particles.size() * sizeof(Particle);
    hipMalloc((void**)&d_particles, particlesSize);
    hipMemcpy(d_particles, particles.data(), particlesSize, hipMemcpyHostToDevice);

    // Transfer particle masses to constant memory
    float* d_masses;
    size_t particleMassesSize = particles.size() * sizeof(float);
    hipMalloc((void**)&d_masses, particleMassesSize);
    std::vector<float> particle_masses(particles.size());
    for (int i = 0; i < particles.size(); ++i) {
        particle_masses[i] = particles[i].mass;
    }
    hipMemcpy(d_masses, particle_masses.data(), particleMassesSize, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 1024;
    int gridSize = (particles.size() + blockSize - 1) / blockSize;

    int sharedMemSize = blockSize * sizeof(Particle);
    simulate_particles_cuda_kernel<<<gridSize, blockSize, sharedMemSize>>>(d_particles, d_masses, G_CONST, dt, particles.size());

    // Transfer particles back to host memory
    hipMemcpy(particles.data(), d_particles, particlesSize, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_particles);
    hipFree(d_masses);

    // End timer
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<float> duration = end - start;
    float seconds = duration.count();

    std::vector<float> system_momentum = calculate_system_momentum(particles);
    float system_energy = calculate_system_energy(particles, G_CONST);

    std::cout << "Verlet_velocity Time: " << (num_steps) * dt << std::endl;
    std::cout << "System Momentum (X, Y): (" << system_momentum[0] << ", " << system_momentum[1] << ")" << std::endl;
    std::cout << "System Energy: " << system_energy << std::endl;
    std::cout << "Particles: " << particles.size() << std::endl;
    // Output the runtime
     printf("Runtime: %f seconds\n", seconds);


    return 0;
}
