#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>
#include <omp.h>

// Define particle structure
struct Particle {
    float posi[2];
    float velocity[2];
    float acceleration[2];
    float mass;
};

// Calculate system momentum
std::vector<float> calculate_system_momentum(const std::vector<Particle>& particles) {
    float total_momentum_x = 0.0;
    float total_momentum_y = 0.0;

    for (const Particle& p : particles) {
        total_momentum_x += p.mass * p.velocity[0];
        total_momentum_y += p.mass * p.velocity[1];
    }

    return { total_momentum_x, total_momentum_y };
}

// Calculate system energy
float calculate_system_energy(const std::vector<Particle>& particles, float G) {
    float total_energy = 0.0;

    for (const Particle& p : particles) {
        float kinetic_energy = 0.5 * p.mass * (p.velocity[0] * p.velocity[0] + p.velocity[1] * p.velocity[1]);
        float potential_energy = 0.0;

        for (const Particle& other : particles) {
            if (&p != &other) {
                float dx = other.posi[0] - p.posi[0];
                float dy = other.posi[1] - p.posi[1];
                float dist = sqrt(dx * dx + dy * dy);
                potential_energy -= G * p.mass * other.mass / dist;
            }
        }

        total_energy += kinetic_energy + potential_energy;
    }

    return total_energy;
}

void calculate_gravity(std::vector<Particle>& particles, float G) {
    for (Particle& p1 : particles) {
        for (Particle& p2 : particles) {
            if (&p1 != &p2) {
                float dx = p2.posi[0] - p1.posi[0];
                float dy = p2.posi[1] - p1.posi[1];
                float dist_squared = dx * dx + dy * dy;
                float dist_cubed = dist_squared * sqrt(dist_squared);

                float force_magnitude = G * p1.mass * p2.mass / dist_cubed;
                float force_x = force_magnitude * dx;
                float force_y = force_magnitude * dy;

                p1.acceleration[0] += force_x / p1.mass;
                p1.acceleration[1] += force_y / p1.mass;
            }
        }
    }
}

void Verlet_velocity(std::vector<Particle>& particles, float G, float dt) {
    for (Particle& p : particles) {
        p.velocity[0] += p.acceleration[0] * dt;
        p.velocity[1] += p.acceleration[1] * dt;

        p.posi[0] += p.velocity[0] * dt;
        p.posi[1] += p.velocity[1] * dt;

        p.acceleration[0] = 0.0;
        p.acceleration[1] = 0.0;
    }
}

__global__ void calculate_gravity_cuda_kernel(Particle* particles, float G, int num_particles) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p1 = particles[index];

        for (int j = 0; j < num_particles; j++) {
            if (j != index) {
                Particle& p2 = particles[j];
                float dx = p2.posi[0] - p1.posi[0];
                float dy = p2.posi[1] - p1.posi[1];
                float dist_squared = dx * dx + dy * dy;
                float dist_cubed = dist_squared * sqrt(dist_squared);

                float force_magnitude = G * p1.mass * p2.mass / dist_cubed;
                float force_x = force_magnitude * dx;
                float force_y = force_magnitude * dy;

                atomicAdd(&p1.acceleration[0], force_x / p1.mass);
                atomicAdd(&p1.acceleration[1], force_y / p1.mass);


            }
        }
    }
}

__global__ void Verlet_velocity_cuda_kernel(Particle* particles, float G, float dt, int num_particles) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p = particles[index];

        p.velocity[0] += p.acceleration[0] * dt;
        p.velocity[1] += p.acceleration[1] * dt;

        p.posi[0] += p.velocity[0] * dt;
        p.posi[1] += p.velocity[1] * dt;

        p.acceleration[0] = 0.0;
        p.acceleration[1] = 0.0;
    }
}

int main() {
    // Define simulation parameters
    const float G = 6.674e-11;
    const float M_PI = 3.14159;

    std::cout << std::fixed << std::setprecision(12);

    // Input time and time step
    float t = 4.0 * M_PI / sqrt(G * 1000000000);
    float dt = 0.001;

    // Perform simulation
    int num_steps = t / dt;

    std::vector<Particle> particles = {
    { {0.0f, 1.0f}, {sqrtf(G*1000000000.0f)/2.0f, 0.0f}, {0.0f, 0.0f}, 1000000000.0f },
    { {0.0f, -1.0f}, {-sqrtf(G*1000000000.0f)/2.0f, 0.0f}, {0.0f, 0.0f}, 1000000000.0f }
    };


    float start_time = omp_get_wtime();

    // Perform Verlet simulation using CUDA
    Particle* d_particles;
    size_t particlesSize = particles.size() * sizeof(Particle);

    hipMalloc((void**)&d_particles, particlesSize);
    hipMemcpy(d_particles, particles.data(), particlesSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (particles.size() + blockSize - 1) / blockSize;

    for (int i = 0; i <= num_steps; i++) {
        calculate_gravity_cuda_kernel<<<gridSize, blockSize>>>(d_particles, G, particles.size());
        Verlet_velocity_cuda_kernel<<<gridSize, blockSize>>>(d_particles, G, dt, particles.size());
    }

    hipMemcpy(particles.data(), d_particles, particlesSize, hipMemcpyDeviceToHost);
    hipFree(d_particles);

    float end_time = omp_get_wtime();

    std::vector<float> system_momentum = calculate_system_momentum(particles);
    float system_energy = calculate_system_energy(particles, G);

    std::cout << "Verlet_velocity Time: " << (num_steps) * dt << std::endl;
    std::cout << "System Momentum (X, Y): (" << system_momentum[0] << ", " << system_momentum[1] << ")" << std::endl;
    std::cout << "System Energy: " << system_energy << std::endl;
    std::cout << "Total Execution Time: " << end_time - start_time << " seconds" << std::endl;

    return 0;
}
