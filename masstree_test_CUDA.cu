#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>
#include <omp.h>
#include <random>
#include <chrono>


// Define particle structure
struct Particle {
    float posi[2];
    float velocity[2];
    float acceleration[2];
    float acceleration_prev[2];
    float mass;
};

// Calculate system momentum
std::vector<float> calculate_system_momentum(const std::vector<Particle>& particles) {
    float total_momentum_x = 0.0;
    float total_momentum_y = 0.0;

    for (const Particle& p : particles) {
        total_momentum_x += p.mass * p.velocity[0];
        total_momentum_y += p.mass * p.velocity[1];
    }

    return { total_momentum_x, total_momentum_y };
}

// Calculate system energy
float calculate_system_energy(const std::vector<Particle>& particles, float G) {
    float total_energy = 0.0;

    for (const Particle& p : particles) {
        float kinetic_energy = 0.5 * p.mass * (p.velocity[0] * p.velocity[0] + p.velocity[1] * p.velocity[1]);
        float potential_energy = 0.0;

        for (const Particle& other : particles) {
            if (&p != &other) {
                float dx = other.posi[0] - p.posi[0];
                float dy = other.posi[1] - p.posi[1];
                float dist = sqrt(dx * dx + dy * dy);
                potential_energy -= G * p.mass * other.mass / dist;
            }
        }

        total_energy += kinetic_energy + potential_energy;
    }

    return total_energy;
}



__global__ void simulate_particles_cuda_kernel(Particle* particles, const float* particle_masses, double G, double dt, int num_particles) {
    extern __shared__ Particle sharedParticles[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_particles) {
        Particle& p = particles[index];
        Particle& shared_p = sharedParticles[threadIdx.x];

        shared_p = p;

        __syncthreads();

        for (int j = 0; j < num_particles; j++) {
            if (j != index) {
                Particle& other = sharedParticles[j];
                float dx = other.posi[0] - shared_p.posi[0];
                float dy = other.posi[1] - shared_p.posi[1];
                float dist_squared = dx * dx + dy * dy;
                float dist_cubed = dist_squared * sqrt(dist_squared);

                float force_magnitude = G * shared_p.mass * particle_masses[j] / dist_cubed;
                float force_x = force_magnitude * dx;
                float force_y = force_magnitude * dy;

                atomicAdd(&shared_p.acceleration[0], force_x / shared_p.mass);
                atomicAdd(&shared_p.acceleration[1], force_y / shared_p.mass);
            }
        }

        p.posi[0] += p.velocity[0] * dt + 0.5 * p.acceleration[0] * dt * dt;
        p.posi[1] += p.velocity[1] * dt + 0.5 * p.acceleration[1] * dt * dt;
        p.velocity[0] += 0.5 * (p.acceleration[0] + p.acceleration_prev[0]) * dt;
        p.velocity[1] += 0.5 * (p.acceleration[1] + p.acceleration_prev[1]) * dt;
        p.acceleration_prev[0] = p.acceleration[0];
        p.acceleration_prev[1] = p.acceleration[1];

        p.acceleration[0] = 0.0f;
        p.acceleration[1] = 0.0f;
    }
}


int main() {
    // Define simulation parameters
    const float G = 6.674e-11;
    const float M_PI = 3.14159;

    std::cout << std::fixed << std::setprecision(12);

    // Input time and time step
    float t = 0.001;
    float dt = 0.001;

    // Perform simulation
    int num_steps = t / dt;

    // Random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist_pos(-1.0f, 1.0f);
    std::uniform_real_distribution<float> dist_vel(-10.0f, 10.0f);
    std::uniform_real_distribution<float> dist_mass(1.0e6f, 1.0e7f);

    // Generate particles
    std::vector<Particle> particles;
    particles.reserve(131072);
    for (int i = 0; i < 131072; ++i) {
        Particle particle;
        particle.posi[0] = dist_pos(gen);
        particle.posi[1] = dist_pos(gen);
        particle.velocity[0] = dist_vel(gen);
        particle.velocity[1] = dist_vel(gen);
        particle.acceleration[0] = 0.0f;
        particle.acceleration[1] = 0.0f;
        particle.mass = dist_mass(gen);
        particles.push_back(particle);
    }

    // Start timer
    auto start = std::chrono::high_resolution_clock::now();

    // Transfer particles to device memory
    Particle* d_particles;
    size_t particlesSize = particles.size() * sizeof(Particle);
    hipMalloc((void**)&d_particles, particlesSize);
    hipMemcpy(d_particles, particles.data(), particlesSize, hipMemcpyHostToDevice);

    // Transfer particle masses to constant memory
    float* d_masses;
    size_t particleMassesSize = particles.size() * sizeof(float);
    hipMalloc((void**)&d_masses, particleMassesSize);
    std::vector<float> particle_masses(particles.size());
    for (int i = 0; i < particles.size(); ++i) {
        particle_masses[i] = particles[i].mass;
    }
    hipMemcpy(d_masses, particle_masses.data(), particleMassesSize, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 1024;
    int gridSize = (particles.size() + blockSize - 1) / blockSize;

    int sharedMemSize = blockSize * sizeof(Particle);
    simulate_particles_cuda_kernel<<<gridSize, blockSize, sharedMemSize>>>(d_particles, d_masses, G, dt, particles.size());

    // Transfer particles back to host memory
    hipMemcpy(particles.data(), d_particles, particlesSize, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_particles);
    hipFree(d_masses);

    // End timer
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> duration = end - start;
    double seconds = duration.count();

    std::vector<float> system_momentum = calculate_system_momentum(particles);
    float system_energy = calculate_system_energy(particles, G);

    std::cout << "Verlet_velocity Time: " << (num_steps) * dt << std::endl;
    std::cout << "System Momentum (X, Y): (" << system_momentum[0] << ", " << system_momentum[1] << ")" << std::endl;
    std::cout << "System Energy: " << system_energy << std::endl;
    std::cout << "Particles: " << particles.size() << std::endl;
    // Output the runtime
     printf("Runtime: %f seconds\n", seconds);


    return 0;
}

